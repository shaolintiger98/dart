#include "hip/hip_runtime.h"
#include "kinematics.h"

#include <cmath>
#include <ctime>

#include "kernel_common.h"
#include "geometry/grid_3d.h"
#include "geometry/SE3.h"
#include "optimization/optimization.h"
#include "util/mirrored_memory.h"

namespace dart {

static const float truncVal = 1000.0;


// -=-=-=-=-=-=-=-=-=- kernels -=-=-=-=-=-=-=-=-=-
// probably need some well defined place to store the result
// that we can access later
// also a flag or something to show when we're done that the main
// thread can block on
// use hipMemcpy(_hVector,_dVector,_length*sizeof(T),hipMemcpyDeviceToHost); for getting things back
__global__ void gpu_computeForwardKinematics(
        const int maxidx,
        const int numtoproc,
        const int poselen,
        // array that corresponds to a pose/joint configuration
        const float * poses,
        //const int numDofs,
        // result store
        SE3 *T_mfss,
        //const int numJoints,
        const float2 * jointLims,
        const SE3 *T_pfs,
        const int *frameParents,
        const int numframes,
        const JointType *jointTypes,
        const float3 *jointAxes) {
    int start_idx = (threadIdx.x + blockIdx.x * blockDim.x) * numtoproc;
    for (int idx = start_idx;idx < start_idx + numtoproc; ++idx){
    if (idx >= maxidx) return;
    const float * pose = poses + poselen * idx;
    SE3 * T_mfs = T_mfss + numframes * idx;
    SE3 val;
    // setup
    T_mfs[0] = val;
    /*printf("test\n[%f, %f, %f, %f,\n%f, %f, %f, %f,\n%f, %f, %f, %f]\n", T_mfs[0].r0.x, T_mfs[0].r0.y, T_mfs[0].r0.z, T_mfs[0].r0.w,
           T_mfs[0].r1.x, T_mfs[0].r1.y, T_mfs[0].r1.z, T_mfs[0].r1.w, T_mfs[0].r2.x, T_mfs[0].r2.y, T_mfs[0].r2.z, T_mfs[0].r2.w);*/

    /*for (int i = 0; i < numframes; ++i) {
        printf("[%f, %f, %f, %f,\n%f, %f, %f, %f,\n%f, %f, %f, %f]\n", T_mfs[i].r0.x, T_mfs[i].r0.y, T_mfs[i].r0.z, T_mfs[i].r0.w,
               T_mfs[i].r1.x, T_mfs[i].r1.y, T_mfs[i].r1.z, T_mfs[i].r1.w, T_mfs[i].r2.x, T_mfs[i].r2.y, T_mfs[i].r2.z, T_mfs[i].r2.w);
    }*/// for all of the things in our range
    // create the SE3 off the joint config
    // start forward multiplying the mfs

    /*int numframes = robot.getNumFrames();
    SE3 *T_pfs = robot.getDeviceTransformsParentJointToFrame();
    // place to store the results
    SE3 *T_mfs = robot.getDeviceTransformsFrameToModel();
    int *frameParents = robot.getDeviceFrameParents();void
    //model.h
    // make accessor for _jointLimits
    // joint limits for getjointmin/max or do the min operation before
    // pass in joint limits
    JointType *jointTypes = robot.getDeviceJointTypes();
    float3 * jointAxes = robot.getDeviceJointAxes();*/
    // _jointAxes

    // need to figure out how to turn a configuration into a full set of joint angles (pose reduction to full pose)
    // for now assume that jointConfigurations has already been transformed into a full pose
    int j = 6;
    //printf("I was called:%d\n", numframes);
    for (int f=1; f<numframes; ++f) {

        //printf("1:\n");
                //printf("%d:1\n", jointLims[j].x);
                        //printf("%d:2\n", pose[j]);
                        //printf("%d:3\n", jointLims[j].y);
        float p_ = pose[j];
        if(j >= poselen) p_ = 0.0;
        float p = fminf(fmaxf(jointLims[j-6].x,p_),jointLims[j-6].y);

        const int joint = f-1;
                //printf("1.5:\n");
        SE3 T_pf = T_pfs[joint];
        //printf("2:\n");
        switch(jointTypes[joint]) {
            case RotationalJoint:
                T_pf = T_pf*SE3Fromse3(se3(0, 0, 0,
                                p*jointAxes[joint].x, p*jointAxes[joint].y, p*jointAxes[joint].z));
                ++j;
                break;
            case PrismaticJoint:
                T_pf = T_pf*SE3Fromse3(se3(p*jointAxes[joint].x, p*jointAxes[joint].y, p*jointAxes[joint].z,
                                0, 0, 0));
                ++j;
                break;
        }
                //printf("3:\n");
        const int parent = frameParents[f];
                //printf("4:\n");
        T_mfs[f] = T_mfs[parent]*T_pf;

            //printf("Results:\n");
        //printf("[%f, %f, %f, %f,\n%f, %f, %f, %f,\n%f, %f, %f, %f]\n", T_mfs[f].r0.x, T_mfs[f].r0.y, T_mfs[f].r0.z, T_mfs[f].r0.w,
        //       T_mfs[f].r1.x, T_mfs[f].r1.y, T_mfs[f].r1.z, T_mfs[f].r1.w, T_mfs[f].r2.x, T_mfs[f].r2.y, T_mfs[f].r2.z, T_mfs[f].r2.w);
    }}

    // sync to device - rewrite to sync T_mfs
    //hipMemcpy(T_mfs_host,T_mfs,T_mfs_size,hipMemcpyDeviceToHost);
}

void computeForwardKinematics(
        float *host_pose,
        int poselen,
        MirroredModel &robot,
        MirroredVector<float2> *limits) {
    // limit pose by joint limits and extend the articulation
    // make sure not to leak around this
    //MirroredVector<float2> limits = new MirroredVector<float2>(robot._jointLimits);
    // move these to a spot on the gpu
    // need to tune these/accept larger parameters so that this makes sense
    //dim3 block(8,1,1);
    //dim3 grid(ceil(total / block.x), ceil(total / block.y));
    //std::cout << host_pose[0] << std::endl;
    float *device_pose;
    hipError_t mal_err = hipMalloc((void**) &device_pose, sizeof(float) * poselen);
    //float *posea = (float*) malloc(sizeof(float) * poselen);
    hipError_t mc_err1 = hipMemcpy(device_pose, host_pose, sizeof(float) * poselen, hipMemcpyHostToDevice);
    //hipError_t mc_err2 = hipMemcpy(posea, device_pose, sizeof(float) * poselen, hipMemcpyDeviceToHost);
    if (mal_err != hipSuccess) {
        std::cout << "Cuda malloc error:" << hipGetErrorString(mal_err) << std::endl;
        return;
    }
    if (mc_err1 != hipSuccess) {
        std::cout << "Cuda copy error1:" << hipGetErrorString(mc_err1) << std::endl;
        return;
    }
    /*if (mc_err2 != hipSuccess) {
        std::cout << "Cuda copy error2:" << hipGetErrorString(mc_err2) << std::endl;
        return;
    }
    /*std::cout << "Calling fk code with " << robot.getNumFrames() << " frames, " << poselen << " dimensions" << std::endl;
    for (int i = 0; i < robot.getNumFrames() - 1; ++i) {
        std::cout << "POSEX: " << posea[i] << " " << limits->hostPtr()[i].x
                  << " " << limits->hostPtr()[i].y <<  std::endl;
    }
    free(posea);*/
    robot.syncKinematicsHostToDevice();

    std::cout << 1 <<" " << poselen << " " << (uint64_t) limits->devicePtr() << std::endl;
    gpu_computeForwardKinematics<<<1,1>>>(
                1,1,
                poselen,
                device_pose,
                robot.getDeviceTransformsFrameToModel(),
                limits->devicePtr(),
                robot.getDeviceTransformsParentJointToFrame(),
                robot.getDeviceFrameParents(),
                robot.getNumFrames(),
                robot.getDeviceJointTypes(),
                robot.getDeviceJointAxes()
                );
    hipFree(device_pose);
    robot.syncKinematics();
}

SE3 **computeForwardKinematicsBatch(
        float *host_poses,
        int nposes,
        int ndims,
        MirroredModel &robot,
        MirroredVector<float2> *limits) {

    clock_t s = clock();
    // limit pose by joint limits and extend the articulation
    // make sure not to leak around this
    //MirroredVector<float2> limits = new MirroredVector<float2>(robot._jointLimits);
    // move these to a spot on the gpu
    // need to tune these/accept larger parameters so that this makes sense
    //dim3 block(8,1,1);
    //dim3 grid(ceil(total / block.x), ceil(total / block.y));
    // round up to a power of two for better calculation
    int calc_num = std::pow(2, std::ceil(std::log(nposes)/std::log(2)));
    int nblk = 16;
    int perthr = 128;
    int threadct = max(calc_num / perthr / nblk, 1);
    int nframes = robot.getNumFrames();
    float *device_poses;
    hipError_t mal_err = hipMalloc((void**) &device_poses, sizeof(float) * ndims * nposes);
    if (mal_err != hipSuccess) {
        std::cout << "Cuda malloc error:" << hipGetErrorString(mal_err) << std::endl;
        return NULL;
    }
    SE3 *device_results;
    mal_err = hipMalloc((void**) &device_results, sizeof(SE3) * nframes * nposes);
    if (mal_err != hipSuccess) {
        std::cout << "Cuda malloc error:" << hipGetErrorString(mal_err) << std::endl;
        return NULL;
    }
    hipMemset(device_results,0, sizeof(SE3) * nframes * nposes);

    clock_t e = clock();
    std::cout << "CUDA Mallocs: " << e << " " << s << " " << (e-s) << std::endl;
    s = clock();
    hipError_t mc_err1 = hipMemcpy(device_poses, host_poses, sizeof(float) * ndims * nposes, hipMemcpyHostToDevice);
    //for (int i = 0; i < host_poses.size(); ++i) {
        //hipError_t mc_err1 = hipMemcpy(device_poses + ndims * i, host_poses[i], sizeof(float) * ndims, hipMemcpyHostToDevice);
        if (mc_err1 != hipSuccess) {
            std::cout << "Cuda copy error1:" << hipGetErrorString(mc_err1) << std::endl;
            return NULL;
        }
    //}

    e = clock();
    std::cout << "Device setup: " << e << " " << s << " " << (e-s) << std::endl;
    /*float *posea = (float*) malloc(sizeof(float) * ndims);
    hipError_t mc_err2 = hipMemcpy(posea, device_poses, sizeof(float) * ndims, hipMemcpyDeviceToHost);
    if (mc_err2 != hipSuccess) {
        std::cout << "Cuda copy error2:" << hipGetErrorString(mc_err2) << std::endl;
        return NULL;
    }
    //std::cout << "Calling fk code with " << robot.getNumFrames() << " frames, " << ndims << " dimensions" << std::endl;
    for (int i = 0; i < ndims; ++i) {
        std::cout << "POSEX: " << posea[i] << " " << limits->hostPtr()[i].x
                  << " " << limits->hostPtr()[i].y << " " <<host_poses[i] <<  std::endl;
    }
    free(posea);//*/
    s = clock();
    SE3 **results = (SE3 **) malloc(sizeof(SE3*) * nposes + sizeof(SE3) * nframes * nposes);
    SE3 *datapos = (SE3 *) &results[nposes];
    // initalize raw array
    for (int i = 0; i < nposes; ++i) {
        results[i] = datapos + i * nframes;
    }
    e = clock();
    std::cout << "End Setup: " << e << " " << s << " " << (e-s) << std::endl;

    /*hipMemcpy(datapos, device_results, sizeof(SE3) * nframes * nposes, hipMemcpyDeviceToHost);
    std::cout << "Result " << datapos[2].r0.w << " " << datapos[2].r0.x << " " <<datapos[2].r0.y << " " <<
                 datapos[2].r0.z << " " <<std::endl;
    std::cout << nposes <<" " << ndims << " " << (uint64_t) limits->devicePtr() << std::endl;*/
    s = clock();
    std::cout << "Launching " << calc_num<< " "<< threadct << " " << nblk << " " << perthr << std::endl;
    gpu_computeForwardKinematics<<<threadct,nblk>>>(
                                            nposes,
                                                         perthr,
                                            ndims,
                                            device_poses,
                                            device_results,
                                            limits->devicePtr(),
                                            robot.getDeviceTransformsParentJointToFrame(),
                                            robot.getDeviceFrameParents(),
                                            robot.getNumFrames(),
                                            robot.getDeviceJointTypes(),
                                            robot.getDeviceJointAxes()
                                            );
    e = clock();
    std::cout << "Actual computation: " << e << " " << s << " " << (e-s) << std::endl;

                /*nposes,
                ndims,
                device_poses,
                robot.getDeviceTransformsFrameToModel(),//device_results,
                limits->devicePtr(),
                robot.getDeviceTransformsParentJointToFrame(),
                robot.getDeviceFrameParents(),
                robot.getNumFrames(),
                robot.getDeviceJointTypes(),
                robot.getDeviceJointAxes()
                );*/
    //hipFree(device_poses);

    /*hipMemcpy(datapos, device_results, sizeof(SE3) * nframes * nposes, hipMemcpyDeviceToHost);
    std::cout << "Result " << datapos[2].r0.w << " " << datapos[2].r0.x << " " <<datapos[2].r0.y << " " <<
                 datapos[2].r0.z << " " <<std::endl;

    std::cout << (uint64_t) results << " " << (uint64_t) datapos << " " << (uint64_t) results[0] << " " << datapos[0].r0.w <<std::endl;
    std::cout << results[0][0].r0.w <<" " << datapos[0].r0.w << " " <<std::endl;*/
    s = clock();
    hipMemcpy(datapos, device_results, sizeof(SE3) * nframes * nposes, hipMemcpyDeviceToHost);
    e = clock();
    std::cout << "End Memcpy: " << e << " " << s << " " << (e-s) << std::endl;
    hipFree(device_results);
    return results;
}

}
